#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include<ctime>
#include <stdio.h>
#include <chrono>
#include <stdlib.h>     /* srand, rand */

__global__ void insertion_sort(int* p, int n) {

}
void random_me(int* p, size_t n) {
    srand(time(NULL));
    for (int i = 0; i < n; i++)*(p + i)=(rand()%1111)%100;

}
__global__ void kern(int* p) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index == 420) { p[index] = 69; return; }
    p[index] = 9000;
}

int main()
{
    constexpr int N = 1000;

    int* arr = (int*)malloc(N*sizeof(int));
    random_me(arr, N);
    auto arr_d = arr;
    hipMalloc(&arr_d, N * sizeof(int));
    hipMemcpy(arr_d,arr_d,N*sizeof(int),hipMemcpyHostToDevice);
    kern << <10,100  >> > (arr_d);
    hipDeviceSynchronize();
    hipMemcpy(arr, arr_d, N * sizeof(int), hipMemcpyDeviceToHost);

    printf("%d", arr[420]);
}


