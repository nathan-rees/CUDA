#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include<ctime>
#include <stdio.h>
#include <chrono>
#include <stdlib.h>     /* srand, rand */
#include <math.h>       /* log2 */

__global__ void merge(int* ptr,size_t size) {
    int i = threadIdx.x;
    int sort[32];//tempi think this is using slow memory
    int* start = ptr + (i * size);
    int* mid = start + (size / 2);
    int a = 0; int b = 0;
    for (int j = 0; j < size; j++) {
        if (a > (size / 2)-1) { a--; start[a] = 9999; }
        else if (b > (size / 2)-1) 
        { b--; mid[b] = 9999; }//i dont know what to say

        if (start[a] > mid[b]) {
            sort[j] = mid[b];
            b++;
        }

        else {
            sort[j] = start[a];
            a++;
        }
        
    }
          
    start = ptr + (i * size);      
    for (int j = 0; j < size; j++) {          
        start[j] =  sort[j];

    }

}

void random_me(int* p, size_t n) {
    srand(time(NULL));
    for (int i = 0; i < n; i++)*(p + i)=(rand()%1111)%100;
}


int main()
{
    constexpr int N = 32;

    int* arr = (int*)malloc(N * sizeof(int));
    random_me(arr, N);
    for (int c = 0; c < N; c++) { printf("%d,", arr[c]);  }printf("\n");
    auto arr_d = arr;
    
    hipMalloc(&arr_d, N * sizeof(int));
    hipMemcpy(arr_d, arr, N * sizeof(int), hipMemcpyHostToDevice);
    for (int size = 2; size <N+1; size = size * 2) {
        merge << <1, N/size >> > (arr_d, size);
        hipDeviceSynchronize();

    }
    
    
    hipMemcpy(arr, arr_d, N * sizeof(int), hipMemcpyDeviceToHost);
    for (int c = 0; c < N;c++) {
        printf("%d\n", arr[c]);
    }
    
}


